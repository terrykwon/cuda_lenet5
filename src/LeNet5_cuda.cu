#include "hip/hip_runtime.h"
#include "LeNet5_cuda.h"


/**
 * Tiled matrix multiplication code from textbook
 */
// __global__
// void MatrixMulKernel(float* d_M, float* d_N, float* d_P, int Width) {
//   __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
//   __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

//   int bx = blockIdx.x;
//   int by = blockIdx.y;
//   int tx = threadIdx.x;
//   int ty = threadIdx.y;

//   int Row = by * TILE_WIDTH + ty;
//   int Col = bx * TILE_WIDTH + tx;

//   float Pvalue = 0;

//   for (int ph = 0; ph < Width/TILE_WIDTH; ++ph) {
//     // collaborative loading of d_M and d_N tiles into shared memory
//     // important! should understand
//     Mds[ty][tx] = d_M[Row*Width + ph*TILE_WIDTH * tx];
//     Nds[ty][tx] = d_N[(ph*TILE_WIDTH + ty)*Width + Col];
//     __syncthreads();

//     for (int k = 0; k < TILE_WIDTH; k++) {
//       Pvalue += Mds[ty][k] * Nds[k][tx];
//     }
//     __syncthreads(); // why?
//   }
//   d_P[Row*Width + Col] = Pvalue;
// }


/**
 * Wrapper to catch CUDA errors.
 * For debugging only.
 */
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

LeNet5_cuda::LeNet5_cuda(int batch) : LeNet5(batch) {
  this->f_conv1_weight = new float[conv1_in_channel * conv1_out_channel *
                                  conv1_kernel_size * conv1_kernel_size];
  this->f_conv1_bias = new float[conv1_out_channel];
  this->f_conv2_weight = new float[conv2_in_channel * conv2_out_channel *
                                  conv2_kernel_size * conv2_kernel_size];
  this->f_conv2_bias = new float[conv2_out_channel];
  this->f_fc1_weight = new float[fc1_in_channel * fc1_out_channel];
  this->f_fc1_bias = new float[fc1_out_channel];
  this->f_fc2_weight = new float[fc2_in_channel * fc2_out_channel];
  this->f_fc2_bias = new float[fc2_out_channel];
  this->f_fc3_weight = new float[fc3_in_channel * fc3_out_channel];
  this->f_fc3_bias = new float[fc3_out_channel];

  // Activation
  this->f_output = new float[batch * output_size];
}

__global__ void conv1(float* input, float* output,
                      int B, int H, int W, int IC, int OC,
                      int K) {
  int H_OUT = H - (K - 1); // output dimensions
  int W_OUT = W - (K - 1);

  int b = blockIdx.x; // batch
  int oc = blockIdx.y; // output channel
  int w = threadIdx.x; // col
  int h = threadIdx.y; // row

  // Convolution
  int output_index =
      b * (OC * H_OUT * W_OUT) + oc * (H_OUT * W_OUT) + h * W_OUT + w;
  output[output_index] = d_conv1_bias[oc];
  for (int ic = 0; ic < IC; ic++) {
    int input_base = b * (IC * H * W) + ic * (H * W) + h * (W) + w;
    int kernel_base = oc * (IC * K * K) + ic * (K * K);
    for (int kh = 0; kh < K; kh++) {
      for (int kw = 0; kw < K; kw++) {
        float val = input[input_base + kh * (W) + kw] *
                      d_conv1_weight[kernel_base + kh * (K) + kw];
        output[output_index] += val;
      }
    }
  }
}

__global__ void conv2(float* input, float* output,
                      int B, int H, int W, int IC, int OC,
                      int K) {
  int H_OUT = H - (K - 1); // output dimensions
  int W_OUT = W - (K - 1);

  int b = blockIdx.x; // batch
  int oc = blockIdx.y; // output channel
  int w = threadIdx.x; // col
  int h = threadIdx.y; // row

  // Convolution
  int output_index =
      b * (OC * H_OUT * W_OUT) + oc * (H_OUT * W_OUT) + h * W_OUT + w;
  output[output_index] = d_conv2_bias[oc];
  for (int ic = 0; ic < IC; ic++) {
    int input_base = b * (IC * H * W) + ic * (H * W) + h * (W) + w;
    int kernel_base = oc * (IC * K * K) + ic * (K * K);
    for (int kh = 0; kh < K; kh++) {
      for (int kw = 0; kw < K; kw++) {
        float val = input[input_base + kh * (W) + kw] *
                      d_conv2_weight[kernel_base + kh * (K) + kw];
        output[output_index] += val;
      }
    }
  }
}


/**
 * (batch_size, 3, 1) x (32, 32, 1)
 */
__global__ void normalize(int batch, int input_channel, int input_size, const uint8_t* const d_image, float* d_input) {
  // automatically placed in registers. Should these be in shared / constant?
  // probably not because they're just single variables
  const float max_int = 255.0f;   
  const float mean = 0.5f;
  const float var = 0.5f;

  const int batch_id = blockIdx.x;
  const int channel_id = blockIdx.y;
  const int col = threadIdx.x;
  const int row = threadIdx.y;

  float val;

  if (col < input_size && row < input_size) {
    // standard normalize, center at 0
    // one global memory read, one write
    val = d_image[batch_id*input_channel*input_size*input_size + channel_id*input_size*input_size + row*input_size + col];
    val = ((val/max_int) - mean) / var;
    d_input[batch_id*input_channel*input_size*input_size + channel_id*input_size*input_size + row*input_size + col] = val;
  }
}

/**
 * (batch_size, out_channels, 1) x (width, height, 1)
 */
__global__ void naive_conv(float* input, float* output, float* weight,
                      float* bias, int B, int H, int W, int IC, int OC,
                      int K) {
  int H_OUT = H - (K - 1); // output dimensions
  int W_OUT = W - (K - 1);

  int b = blockIdx.x; // batch
  int oc = blockIdx.y; // output channel
  int w = threadIdx.x; // col
  int h = threadIdx.y; // row

  // Convolution
  int output_index =
      b * (OC * H_OUT * W_OUT) + oc * (H_OUT * W_OUT) + h * W_OUT + w;
  output[output_index] = bias[oc];
  for (int ic = 0; ic < IC; ic++) {
    int input_base = b * (IC * H * W) + ic * (H * W) + h * (W) + w;
    int kernel_base = oc * (IC * K * K) + ic * (K * K);
    for (int kh = 0; kh < K; kh++) {
      for (int kw = 0; kw < K; kw++) {
        float val = input[input_base + kh * (W) + kw] *
                      weight[kernel_base + kh * (K) + kw];
        output[output_index] += val;
      }
    }
  }
}

/**
 * (batch_size, in_channels, 1) x (width, height, 1)
 * Element-wise. 
 */
__global__ void naive_relu(float* feature_map, int channels, int width, int height) {
  int b = blockIdx.x; // batch
  int oc = blockIdx.y; // output channel
  int w = threadIdx.x; // col
  int h = threadIdx.y; // row

  int index = b * (channels * width * height) + oc * (width * height) + h * width + w;

  feature_map[index] = fmax(feature_map[index], 0.0f);
}

/**
 * (batch_size, in_channels, 1) x (width, height, 1)
 * This is actually much slower than the CPU version.
 */
__global__ void naive_pool(float* input, float* output, int C, int H, int W) {
  int scale = 2;
  int H_OUT = H / scale;
  int W_OUT = W / scale;

  int b = blockIdx.x; // batch
  int c = blockIdx.y; // output channel
  int w = threadIdx.x; // col
  int h = threadIdx.y; // row

  int input_base = b * (C * H * W) + c * (H * W) + (h*2) * (W) + (w*2);
  int max_sh = 0;
  int max_sw = 0;
  float max_val = 0.0f; // since after relu

  // Find maximum
  for (int sh = 0; sh < scale; sh++) {
    for (int sw = 0; sw < scale; sw++) {
      float val = input[input_base + sh * (W) + sw];
      if (val > max_val) {
        max_val = val;
        // max_sh = sh;
        // max_sw = sw;
      }
    }
  }

  // Set output with max value
  int output_index = b * (C * H_OUT * W_OUT) + c * (H_OUT * W_OUT) +
                      h * (W_OUT) + w;
  output[output_index] = max_val;
}

/**
 * (batch, 1, 1) x (output_nodes, 1, 1)
 */
__global__ void naive_fc(float* input, float* output, float* weight, float* bias,
                         int IC, int OC) {
  int b = blockIdx.x;
  int oc = threadIdx.x; 

  output[b * OC + oc] = bias[oc];
  for (int ic = 0; ic < IC; ic++)
    output[b * OC + oc] += weight[oc * IC + ic] * input[b * IC + ic];
}

void LeNet5_cuda::predict(int batch) {
  std::cout << "batch " << batch << std::endl;

  // cpu_normalize(image, input);
  dim3 normGridDim(batch, 3, 1);
  dim3 normBlockDim(32, 32, 1);
  normalize<<<normGridDim, normBlockDim>>>(batch, input_channel, input_size, d_image, d_input);

  // dest, source, ...
  // hipMemcpy(input, d_input, sizeof(float)*batch*input_channel*input_size*input_size, hipMemcpyDeviceToHost);

  // Conv2d
  // cpu_conv(input, C1_feature_map, conv1_weight, conv1_bias, batch, input_size,
  //      input_size, conv1_in_channel, conv1_out_channel, conv1_kernel_size);
  dim3 conv1GridDim(batch, 6, 1);
  dim3 conv1BlockDim(28, 28, 1);
  conv1<<<conv1GridDim, conv1BlockDim>>>(d_input, d_C1_feature_map, batch, input_size,
                                              input_size, conv1_in_channel, conv1_out_channel, conv1_kernel_size);

  // hipMemcpy(C1_feature_map, d_C1_feature_map, sizeof(float)*batch*conv1_out_channel*C1_size*C1_size, hipMemcpyDeviceToHost);

  // cpu_relu(C1_feature_map, batch * C1_channel * C1_size * C1_size);
  dim3 relu1GridDim(batch, 6, 1);
  dim3 relu1BlockDim(28, 28, 1);
  naive_relu<<<relu1GridDim, relu1BlockDim>>>(d_C1_feature_map, C1_channel, C1_size, C1_size);
  // hipMemcpy(C1_feature_map, d_C1_feature_map, sizeof(float)*batch*conv1_out_channel*C1_size*C1_size, hipMemcpyDeviceToHost);

  // MaxPool2d
  // cpu_pool(C1_feature_map, S2_feature_map, batch, C1_channel, C1_size, C1_size);
  dim3 pool1GridDim(batch, 6, 1);
  dim3 pool1BlockDim(14, 14, 1);
  naive_pool<<<pool1GridDim, pool1BlockDim>>>(d_C1_feature_map, d_S2_feature_map, C1_channel, C1_size, C1_size);
  // hipMemcpy(S2_feature_map, d_S2_feature_map, sizeof(float)*batch*conv1_out_channel*S2_size*S2_size, hipMemcpyDeviceToHost);

  // Conv2d
  // cpu_conv(S2_feature_map, C3_feature_map, conv2_weight, conv2_bias, batch, S2_size,
  //      S2_size, conv2_in_channel, conv2_out_channel, conv2_kernel_size);
  dim3 conv2GridDim(batch, 16, 1);
  dim3 conv2BlockDim(10, 10, 1); // too few threads?
  conv2<<<conv2GridDim, conv2BlockDim>>>(d_S2_feature_map, d_C3_feature_map,
      batch, S2_size, S2_size, conv2_in_channel, conv2_out_channel, conv2_kernel_size);

  // cpu_relu(C3_feature_map, batch * C3_channel * C3_size * C3_size);
  dim3 relu2GridDim(batch, 16, 1);
  dim3 relu2BlockDim(10, 10, 1);
  naive_relu<<<relu2GridDim, relu2BlockDim>>>(d_C3_feature_map, C3_channel, C3_size, C3_size); 

  // MaxPool2d
  // cpu_pool(C3_feature_map, S4_feature_map, batch, C3_channel, C3_size, C3_size);
  dim3 pool2GridDim(batch, 16, 1);
  dim3 pool2BlockDim(5, 5, 1);
  naive_pool<<<pool2GridDim, pool2BlockDim>>>(d_C3_feature_map, d_S4_feature_map, C3_channel, C3_size, C3_size);
  // hipMemcpy(S4_feature_map, d_S4_feature_map, sizeof(float)*batch*conv2_out_channel*S4_size*S4_size, hipMemcpyDeviceToHost);

  // Linear
  // cpu_fc(S4_feature_map, C5_layer, fc1_weight, fc1_bias, batch, fc1_in_channel,
  //    fc1_out_channel);
  dim3 fc1GridDim(batch, 1, 1);
  dim3 fc1BlockDim(120, 1, 1);
  naive_fc<<<fc1GridDim, fc1BlockDim>>>(d_S4_feature_map, d_C5_layer, d_fc1_weight, d_fc1_bias, 
      fc1_in_channel, fc1_out_channel);

  // cpu_relu(C5_layer, batch * C5_size);
  dim3 relu3GridDim(batch, 1, 1);
  dim3 relu3BlockDim(120, 1, 1);
  naive_relu<<<relu3GridDim, relu3BlockDim>>>(d_C5_layer, 1, 120, 1); 

  // Linear
  // cpu_fc(C5_layer, F6_layer, fc2_weight, fc2_bias, batch, fc2_in_channel,
  //    fc2_out_channel);
  dim3 fc2GridDim(batch, 1, 1);
  dim3 fc2BlockDim(84, 1, 1);
  naive_fc<<<fc2GridDim, fc2BlockDim>>>(d_C5_layer, d_F6_layer, d_fc2_weight, d_fc2_bias, 
      fc2_in_channel, fc2_out_channel);

  // cpu_relu(F6_layer, batch * F6_size);
  dim3 relu4GridDim(batch, 1, 1);
  dim3 relu4BlockDim(84, 1, 1);
  naive_relu<<<relu4GridDim, relu4BlockDim>>>(d_F6_layer, 1, 84, 1); 

  // Linear
  // cpu_fc(F6_layer, output, fc3_weight, fc3_bias, batch, fc3_in_channel,
  //    fc3_out_channel);
  dim3 fc3GridDim(batch, 1, 1);
  dim3 fc3BlockDim(10, 1, 1);
  naive_fc<<<fc3GridDim, fc3BlockDim>>>(d_F6_layer, d_output, d_fc3_weight, d_fc3_bias, 
      fc3_in_channel, fc3_out_channel);

  // dest, source, number of bytes, transfer type
  // hipMemcpy(d_output, output, sizeof(float)*batch*output_size, hipMemcpyHostToDevice);

  /* NOTE: unless you want to make a major change to this class structure, 
  *  you need to write your output to the device memory d_output 
  *  so that classify() can handle the rest.
  */
}

void LeNet5_cuda::prepare_device_memory(uint8_t* image) {
  // Store all double arrays as floats...
  // std::cout << "Copying as floats" << std::endl;
  std::copy(this->conv1_weight, 
            this->conv1_weight+conv1_in_channel*conv1_out_channel*conv1_kernel_size*conv1_kernel_size,
            this->f_conv1_weight);
  std::copy(this->conv1_bias,
            this->conv1_bias+conv1_out_channel,
            this->f_conv1_bias);
  std::copy(this->conv2_weight,
            this->conv2_weight+conv2_in_channel*conv2_out_channel*conv2_kernel_size*conv2_kernel_size,
            this->f_conv2_weight);
  std::copy(this->conv2_bias,
            this->conv2_bias+conv2_out_channel,
            this->f_conv2_bias);
  std::copy(this->fc1_weight,
            this->fc1_weight+fc1_in_channel*fc1_out_channel,
            this->f_fc1_weight);
  std::copy(this->fc1_bias,
            this->fc1_bias+fc1_out_channel,
            this->f_fc1_bias);
  std::copy(this->fc2_weight,
            this->fc2_weight+fc2_in_channel*fc2_out_channel,
            this->f_fc2_weight);
  std::copy(this->fc2_bias,
            this->fc2_bias+fc2_out_channel,
            this->f_fc2_bias);
  std::copy(this->fc3_weight,
            this->fc3_weight+fc3_in_channel*fc3_out_channel,
            this->f_fc3_weight);
  std::copy(this->fc3_bias,
            this->fc3_bias+fc3_out_channel,
            this->f_fc3_bias);

  // Alloc Model Parameters
  // hipMalloc((void**)&d_conv1_weight,
  //            sizeof(float) * conv1_in_channel * conv1_out_channel *
  //                conv1_kernel_size * conv1_kernel_size);
  // hipMalloc((void**)&d_conv1_bias, sizeof(float) * conv1_out_channel);
  // hipMalloc((void**)&d_conv2_weight,
  //            sizeof(float) * conv2_in_channel * conv2_out_channel *
  //                conv2_kernel_size * conv2_kernel_size);
  // hipMalloc((void**)&d_conv2_bias, sizeof(float) * conv2_out_channel);
  hipMalloc((void**)&d_fc1_weight,
             sizeof(float) * fc1_in_channel * fc1_out_channel);
  hipMalloc((void**)&d_fc1_bias, sizeof(float) * fc1_out_channel);
  hipMalloc((void**)&d_fc2_weight,
             sizeof(float) * fc2_in_channel * fc2_out_channel);
  hipMalloc((void**)&d_fc2_bias, sizeof(float) * fc2_out_channel);
  hipMalloc((void**)&d_fc3_weight,
             sizeof(float) * fc3_in_channel * fc3_out_channel);
  hipMalloc((void**)&d_fc3_bias, sizeof(float) * fc3_out_channel);

  // Alloc Activations
  hipMalloc((void**)&d_image,
             sizeof(uint8_t) * batch * input_size * input_size * input_channel);
  hipMalloc((void**)&d_input,
             sizeof(float) * batch * input_channel * input_size * input_size);
  hipMalloc((void**)&d_C1_feature_map,
             sizeof(float) * batch * C1_channel * C1_size * C1_size);
  hipMalloc((void**)&d_S2_feature_map,
             sizeof(float) * batch * S2_channel * S2_size * S2_size);
  hipMalloc((void**)&d_C3_feature_map,
             sizeof(float) * batch * C3_channel * C3_size * C3_size);
  hipMalloc((void**)&d_S4_feature_map,
             sizeof(float) * batch * S4_channel * S4_size * S4_size);
  hipMalloc((void**)&d_C5_layer, sizeof(float) * batch * C5_size);
  hipMalloc((void**)&d_F6_layer, sizeof(float) * batch * F6_size);
  hipMalloc((void**)&d_output, sizeof(float) * batch * output_size);

  // Copy Parameters

  // hipMemcpy(d_conv1_weight, f_conv1_weight,
  //            sizeof(float) * conv1_in_channel * conv1_out_channel *
  //                conv1_kernel_size * conv1_kernel_size,
  //            hipMemcpyHostToDevice);
  // hipMemcpy(d_conv1_bias, f_conv1_bias, sizeof(float) * conv1_out_channel,
  //            hipMemcpyHostToDevice);
  // hipMemcpy(d_conv2_weight, f_conv2_weight,
  //            sizeof(float) * conv2_in_channel * conv2_out_channel *
  //                conv2_kernel_size * conv2_kernel_size,
  //            hipMemcpyHostToDevice);
  // hipMemcpy(d_conv2_bias, f_conv2_bias, sizeof(float) * conv2_out_channel,
  //            hipMemcpyHostToDevice);

  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_conv1_weight), f_conv1_weight,
             sizeof(float) * conv1_in_channel * conv1_out_channel *
                 conv1_kernel_size * conv1_kernel_size));
  hipMemcpyToSymbol(HIP_SYMBOL(d_conv1_bias), f_conv1_bias, sizeof(float) * conv1_out_channel);
  hipMemcpyToSymbol(HIP_SYMBOL(d_conv2_weight), f_conv2_weight,
             sizeof(float) * conv2_in_channel * conv2_out_channel *
                 conv2_kernel_size * conv2_kernel_size);
  hipMemcpyToSymbol(HIP_SYMBOL(d_conv2_bias), f_conv2_bias, sizeof(float) * conv2_out_channel);

  hipMemcpy(d_fc1_weight, f_fc1_weight,
             sizeof(float) * fc1_in_channel * fc1_out_channel,
             hipMemcpyHostToDevice);
  hipMemcpy(d_fc1_bias, f_fc1_bias, sizeof(float) * fc1_out_channel,
             hipMemcpyHostToDevice);
  hipMemcpy(d_fc2_weight, f_fc2_weight,
             sizeof(float) * fc2_in_channel * fc2_out_channel,
             hipMemcpyHostToDevice);
  hipMemcpy(d_fc2_bias, f_fc2_bias, sizeof(float) * fc2_out_channel,
             hipMemcpyHostToDevice);
  hipMemcpy(d_fc3_weight, f_fc3_weight,
             sizeof(float) * fc3_in_channel * fc3_out_channel,
             hipMemcpyHostToDevice);
  hipMemcpy(d_fc3_bias, f_fc3_bias, sizeof(float) * fc3_out_channel,
             hipMemcpyHostToDevice);
  // copy input image
  size_t image_size = batch * input_size * input_size * input_channel;
  hipMemcpy(d_image, image, image_size * sizeof(uint8_t),
             hipMemcpyHostToDevice);
}

void LeNet5_cuda::classify(int* predict, int batch) {
  // read logits back to cpu
  hipMemcpy(f_output, d_output, sizeof(float) * output_size * batch,
             hipMemcpyDeviceToHost);

  // float back to double
  std::copy(f_output,
            f_output+batch*output_size,
            output);
  // Softmax
  softmax(output, predict, batch, output_size);
}

LeNet5_cuda::~LeNet5_cuda() {
  hipFree(d_conv1_weight);   
  hipFree(d_conv2_weight);   
  hipFree(d_conv1_bias);     
  hipFree(d_conv2_bias);     
  hipFree(d_fc1_weight);     
  hipFree(d_fc2_weight);     
  hipFree(d_fc3_weight);     
  hipFree(d_fc1_bias);       
  hipFree(d_fc2_bias);       
  hipFree(d_fc3_bias);       

  hipFree(d_image);          
  hipFree(d_input);          
  hipFree(d_C1_feature_map); 
  hipFree(d_S2_feature_map); 
  hipFree(d_C3_feature_map); 
  hipFree(d_S4_feature_map); 
  hipFree(d_C5_layer);      
  hipFree(d_F6_layer);     
  hipFree(d_output);       
  hipFree(d_predict_cuda);   
  
  // Free model parameters memories
  delete[] this->f_conv1_weight;
  delete[] this->f_conv1_bias;
  delete[] this->f_conv2_weight;
  delete[] this->f_conv2_bias;
  delete[] this->f_fc1_weight;
  delete[] this->f_fc1_bias;
  delete[] this->f_fc2_weight;
  delete[] this->f_fc2_bias;
  delete[] this->f_fc3_weight;
  delete[] this->f_fc3_bias;
  // // Free activation memories
  // delete[] this->f_input;
  // delete[] this->f_C1_feature_map;
  // delete[] this->f_S2_feature_map;
  // delete[] this->f_C3_feature_map;
  // delete[] this->f_S4_feature_map;
  // delete[] this->f_C5_layer;
  // delete[] this->f_F6_layer;
  delete[] this->f_output;
}


/*** CPU fallbacks ***/

void LeNet5_cuda::cpu_normalize(const uint8_t* const image, float* input) {
  // Initialize variables
  float max_int = 255.0L;
  float mean = 0.5L;
  float var = 0.5L;
  // Normalize
  for (int i = 0; i < batch * input_channel * input_size * input_size; i++) {
    input[i] = image[i] / max_int;       // transforms.ToTensor();
    input[i] = (input[i] - mean) / var;  // transforms.Normalize();
  }
}

void LeNet5_cuda::cpu_relu(float* feature_map, int size) {
  // relu
  for (int i = 0; i < size; i++) feature_map[i] = std::max(feature_map[i], 0.0f);
}

void LeNet5_cuda::cpu_conv(float* input, float* output, float* weight,
                      float* bias, int B, int H, int W, int IC, int OC,
                      int K) {
  // Initialize variable
  int H_OUT = H - (K - 1);
  int W_OUT = W - (K - 1);
  // Convolution
  for (int b = 0; b < B; b++)              // mini-batch
    for (int oc = 0; oc < OC; oc++) {      // Output Channel
      for (int h = 0; h < H_OUT; h++)      // Height
        for (int w = 0; w < W_OUT; w++) {  // Width
          int output_index =
              b * (OC * H_OUT * W_OUT) + oc * (H_OUT * W_OUT) + h * W_OUT + w;
          output[output_index] = bias[oc];
          for (int ic = 0; ic < IC; ic++) {
            int input_base = b * (IC * H * W) + ic * (H * W) + h * (W) + w;
            int kernel_base = oc * (IC * K * K) + ic * (K * K);
            for (int kh = 0; kh < K; kh++)
              for (int kw = 0; kw < K; kw++) {
                float val = input[input_base + kh * (W) + kw] *
                             weight[kernel_base + kh * (K) + kw];
                output[output_index] += val;
              }
          }
        }
    }
}

void LeNet5_cuda::cpu_pool(float* input, float* output, int B, int C, int H,
                      int W) {
  // Initilaize variable
  int scale = 2;
  int H_OUT = H / scale;
  int W_OUT = W / scale;
  // Max Pooling
  for (int b = 0; b < B; b++)
    for (int c = 0; c < C; c++)
      for (int h = 0; h < H; h += 2)
        for (int w = 0; w < W; w += 2) {
          // Init values
          int input_base = b * (C * H * W) + c * (H * W) + h * (W) + w;
          int max_sh = 0;
          int max_sw = 0;
          float max_val = std::numeric_limits<float>::lowest();
          // Find maximum
          for (int sh = 0; sh < scale; sh++)
            for (int sw = 0; sw < scale; sw++) {
              float val = input[input_base + sh * (W) + sw];
              if (val - max_val > std::numeric_limits<float>::epsilon()) {
                max_val = val;
                max_sh = sh;
                max_sw = sw;
              }
            }
          // Set output with max value
          int output_index = b * (C * H_OUT * W_OUT) + c * (H_OUT * W_OUT) +
                             (h / 2) * W_OUT + (w / 2);
          output[output_index] = max_val;
        }
}

void LeNet5_cuda::cpu_fc(float* input, float* output, float* weight, float* bias,
                    int B, int IC, int OC) {
  // Fully Connected
  for (int b = 0; b < B; b++)
    for (int oc = 0; oc < OC; oc++) {
      output[b * OC + oc] = bias[oc];
      for (int ic = 0; ic < IC; ic++)
        output[b * OC + oc] += weight[oc * IC + ic] * input[b * IC + ic];
    }
}
